
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

class Managed {
    public:
        void* operator new(size_t len) {
            void* ptr;
            hipMallocManaged(&ptr, len);
            //cudaDeviceSynchronize();
            return ptr;
        }
        void operator delete(void* ptr) {
            //cudaDeviceSynchronize();
            hipFree(ptr);
        }
};

class Square : public Managed {
    int size;
    public:
        float **arr;
        Square(int size) : size{size} {
            hipMallocManaged(&arr, size*sizeof(float*));
            for (int i = 0; i < size; ++i) {
                hipMallocManaged(&(arr[i]), size*sizeof(float));
                  //std::cout << "right here" << std::endl;
            }
        }
        ~Square() {
            for (int i = 0; i < size; ++i) {
                hipFree(arr[i]);
            }
            hipFree(arr);
        }

        int getSize() {return size;}

        void initialize() {
            for (int i = 0; i < size; ++i) {
                for (int k = 0; k < size; ++k) {
                    arr[i][k] = 1.0f;
                }
            }
        }

        void print() {
            for (int i = 0; i < size; ++i) {
                std::cout << "Row " << i << ": ";
                for (int k = 0; k < size; ++k) {
                    std::cout << arr[i][k] << ", ";
                }
                std::cout << std::endl;
            }
        }
};

__global__
void modifySquare(Square* sqr, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    // each element is multiplied by (column+1), divided by (row+1)

    // grid-stride loop
    for (; row < size; row += (gridDim.y * blockDim.y)) {
        for (; column < size; column += (gridDim.x * blockDim.x)) {
            //sqr->arr[row][column] = (column+1) / (row+1);
            sqr->arr[row][column] = row;
        }
    }
}

int main() {
    int N = 5;
    Square* mySqr = new Square{N};

    // initialize the fields
    mySqr->initialize();

    // launch kernel
    uint threadDim = 16;
    uint blocksDim = (N + threadDim - 1) / N;
    dim3 blocks{blocksDim, blocksDim, 1};
    dim3 threads{threadDim, threadDim, 1};
    
    modifySquare<<<blocks, threads>>>(mySqr, mySqr->getSize());

    hipError_t err = hipGetLastError();
    if (err) {
         std::string myError{hipGetErrorName(err)};
         std::cout << "The following error has been detected: " << myError << std::endl;
    }
    hipDeviceSynchronize();
    
    // print out the result
    mySqr->print();

    delete mySqr;
}