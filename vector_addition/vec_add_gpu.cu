
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x; // total number of threads in grid

  // grid-stride loop
  for (int i = index; i < n; i += stride)
    // seems like stride only used if numBlocks*blockSize cannot fully tile N (hardware limitation?)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  // Allocate Unified Memory -- this means that it is accessible from the CPU and GPU
  float *x;
  float *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize; // rounds up
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}